#define maxNrows				20
#define maxNcols				20
#define DEBUG
//#define DEBUG_SAVE
#define TEMPLATE

#include <iostream>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "Utilities.cuh"
#include "InputOutput.cuh"
#include "TimingGPU.cuh"

#define real_bool	1				// --- 1 is for double, 0 for float
#if (real_bool == 1)
	#define real double
	#define real_MAX DBL_MAX;
#else
	#define real float
	#define real_MAX FLT_MAX;
#endif

#define numGPUs		1

#include "sv.h"

/********/
/* MAIN */
/********/
int main() {

    const unsigned int Nrows			= 6;
	const unsigned int Ncols			= 6;
	const unsigned int numMatrices		= 5000;

	const real		   tol				= 0.0000001;

    const int blockSizeBidiagonalize	= 128;
    const int blockSizeExtractDiagonals	= 128;
    const int blockSizeRearrange		= 128;
    const int blockSizeSturm			= 32;

    srand(time(NULL));

#ifdef DEBUG
	TimingGPU timerGPU;

	double	rearrangeTime				= 0.,
			bidiagonalizationTime		= 0.,
			tridiagAndBisectionTime		= 0.,
			totalTime					= 0.,
			hostToDeviceTime			= 0.,
			totalTimeWithTransfers		= 0.,
			deviceToHostTime			= 0.,
			totalTransfersTime			= 0.,
			totalTimeTest				= 0.;
#endif

    real *inputMatrices; hipHostMalloc(&inputMatrices, Nrows * Ncols * numMatrices * numGPUs * sizeof(real));

#ifdef DEBUG
	real *singularValuesHost; hipHostMalloc(&singularValuesHost, Ncols * numMatrices * numGPUs * sizeof(real));
#endif

    svdPlan<real> plan[numGPUs];
    for (int k = 0; k < numGPUs; k++) createPlan(plan[k], Nrows, Ncols, numMatrices, k);

	const unsigned int numExecutions = 1;
    for (unsigned k = 0; k < numExecutions; k++) {

        // --- Generate random matrices
	    srand(k);
		int N = 5;
	    for (int h = 0; h < Nrows * Ncols * numMatrices * numGPUs; h++)
	            inputMatrices[h]= 2000 * ((double)rand() / (double)(RAND_MAX - 0.2) * (1. / (double)N)) + 100 * ((double)rand() / (double)(RAND_MAX - 0.2) *
				                (1. / (double)N)) + 24;

#ifdef DEBUG_SAVE
		saveCPUrealtxt(inputMatrices, "/home/angelo/cuda-workspace/SVD/Release/inputMatrices.txt", Nrows * Ncols * numMatrices * numGPUs);
#endif

		// --- Compute batched SVD
#ifdef DEBUG
		timerGPU.StartCounter();
#endif
#ifdef TEMPLATE
		my_svd<real, numMatrices, Nrows, Ncols, blockSizeBidiagonalize, blockSizeExtractDiagonals, blockSizeRearrange, blockSizeSturm>(plan, inputMatrices, rearrangeTime, bidiagonalizationTime, tridiagAndBisectionTime, hostToDeviceTime, tol);
#else
		my_svd<real, blockSizeSturm>(plan, inputMatrices, rearrangeTime, bidiagonalizationTime, tridiagAndBisectionTime, hostToDeviceTime, numMatrices, Nrows, Ncols, tol, blockSizeBidiagonalize, blockSizeExtractDiagonals, blockSizeRearrange);
#endif
#ifdef DEBUG
    	totalTimeTest		+= timerGPU.GetCounter();
#endif

#ifdef DEBUG
		timerGPU.StartCounter();
		for (int k = 0; k < numGPUs; k++) {
			gpuErrchk(hipSetDevice(k));
			gpuErrchk(hipMemcpyAsync(singularValuesHost + k * Ncols * numMatrices, plan[k].singularValues, Ncols * numMatrices * sizeof(real), hipMemcpyDeviceToHost));
		}

		deviceToHostTime	+= timerGPU.GetCounter();
#endif
#ifdef DEBUG_SAVE
		saveCPUrealtxt(singularValuesHost, "/home/angelo/cuda-workspace/SVD/Release/singularValues.txt", Ncols * numMatrices * numGPUs);
#endif
	}

    rearrangeTime			/= (numExecutions * numMatrices);
	bidiagonalizationTime	/= (numExecutions * numMatrices);
	tridiagAndBisectionTime	/= (numExecutions * numMatrices);
	hostToDeviceTime		/= (numExecutions * numMatrices);
	deviceToHostTime		/= (numExecutions * numMatrices);
	totalTimeTest			/= (numExecutions * numMatrices);

    totalTimeWithTransfers	= bidiagonalizationTime + tridiagAndBisectionTime + hostToDeviceTime + deviceToHostTime;
    totalTime				= bidiagonalizationTime + tridiagAndBisectionTime;
    totalTransfersTime		= hostToDeviceTime   + deviceToHostTime;

	std::cout << std::scientific << "Nrows \t\t\t\t: " << Nrows << "\n";
	std::cout << std::scientific << "Ncols \t\t\t\t: " << Ncols << "\n";
	std::cout << std::scientific << "numMatrices \t\t\t: " << numMatrices << "\n";
	std::cout << std::scientific << "Rearrange time \t\t\t: " << rearrangeTime << "\n";
	std::cout << std::scientific << "Bidiagonalization time \t\t: " << bidiagonalizationTime << "\n";
	std::cout << std::scientific << "Tridiag and Bisection time \t: " << tridiagAndBisectionTime << "\n";
	std::cout << std::scientific << "Total transfers time \t\t: " << totalTransfersTime << "\n";
	std::cout << std::scientific << "Total time \t\t\t: " << totalTime << "\n";
	std::cout << std::scientific << "Total time with transfers\t: " << totalTimeWithTransfers << "\n";
	std::cout << std::scientific << "Total time test\t\t\t: " << totalTimeTest << "\n";

    for (int k = 0; k < numGPUs; k++) destroyPlan(plan[k], k);

	gpuErrchk(hipDeviceReset());

    std::cout << "finish\n";

    return 0;
}
