#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <iostream>

#include <thrust\device_vector.h>
#include <thrust\transform_reduce.h>
#include <thrust\reduce.h>
#include <thrust\tuple.h>

#include "BBComplex.h"
#include "InputOutput.cuh"
#include "Utilities.cuh"
#include "Matlab_like.cuh"
#include "Polynomials.cuh"
#include "Synthesis.cuh"
#include "NFFT2_2D.cuh"
#include "NDFT2_2D.cuh"

#define pi 3.141592653589793238463

// --- Algorithm parameters
#define freq				((14.25)*(1e9))					// Operating frequency 
#define lambda				((3e8)/(freq))                  // Wavelength
#define beta				((2.*pi)/(lambda))              // Wavenumber

#define M_x					11                              // Number of reflectarray elements along the x-axis
#define M_y					11                              // Number of reflectarray elements along the y-axis

#define dx					((0.5)*(lambda))                // dist elem x (per array)
#define dy					((0.5)*(lambda))                // dist elem y (per array)

#define aap					(((M_x)-(1))*((dx)/(2)))        // Reflectarray semi-dimension along the x-axis
#define bap					(((M_y)-(1)))*((dy)/(2))        // Reflectarray semi-dimension along the y-axis

#define mfact				12                              // Feed pattern: cos^mfact(theta)

// ??? INUTILI ???
#define dmin				((0.51)*(lambda))               // Minimum allowed inter-element spacing
#define dmax				((0.7 )*(lambda))               // Maximum allowed inter-element spacing

#define dmin_x				((0.51)*(lambda))               // Minimum allowed inter-element spacing along the x-axis
#define dmin_y				((0.51)*(lambda))               // Minimum allowed inter-element spacing along the y-axis
#define dmax_x				((0.7 )*(lambda))               // Maximum allowed inter-element spacing along the x-axis
#define dmax_y				((0.7 )*(lambda))               // Maximum allowed inter-element spacing along the y-axis

#define z0					((2)*(0.8)*(sqrt((aap)*(aap)+(bap)*(bap))))
															// Focal length of the reflectarray surface

#define feed_center_x		0.								
#define feed_center_y		((1.15)*(bap))
#define feed_center_z		(-z0)

#define alfa				(-atan((feed_center_y)/(feed_center_z)))            
															// Feed illumination angle

#define Num_unknowns_x		5								// Number of unknowns for the element positions along the x-axis
#define Num_unknowns_y		5								// Number of unknowns for the element positions along the y-axis

#define Num_unknowns_phases	6								// Number of unknowns for the phase representation

#define chi_u_prime			4                               // Spectral oversampling factor along u
#define chi_v_prime			4                               // Spectral oversampling factor along v

#define a_prime				((chi_u_prime)*(aap))			
#define b_prime				((chi_v_prime)*(bap))

#define u_max				((beta)/(2.))					// Maximum value of the spectral region along the u axis
#define u_min				(-(beta)/(2.))					// Minimum value of the spectral region along the u axis
#define v_max				((beta)/(2.))					// Maximum value of the spectral region along the v axis
#define v_min				(-(beta)/(2.))					// Minimum value of the spectral region along the v axis

#define DEBUG

/********/
/* MAIN */
/********/
int main()
{
	hipblasHandle_t handle; cublasSafeCall(hipblasCreate(&handle));
	
	// --- Defining spectral quantities
	int Nu, Nv;
	thrust::pair<thrust::pair<double *, double *>, double *> d_SpectralTuple = defineSpectralQuantities(u_max, v_max, a_prime, b_prime, beta, &Nu, &Nv);
	thrust::pair<double *, double *> d_UV_discrete = d_SpectralTuple.first;
	double *d_U_discrete = d_UV_discrete.first;
	double *d_V_discrete = d_UV_discrete.second;
	double *d_Filter	 = d_SpectralTuple.second;

	saveGPUrealtxt(d_U_discrete,	"C:\\Users\\angelo\\Documents\\CEM\\ParticleSwarm\\ParticleSwarmSynthesis\\ParticleSwarmSynthesisMatlab\\U_discrete.txt", (2 * Nu) * (2 * Nv));
	saveGPUrealtxt(d_V_discrete,	"C:\\Users\\angelo\\Documents\\CEM\\ParticleSwarm\\ParticleSwarmSynthesis\\ParticleSwarmSynthesisMatlab\\V_discrete.txt", (2 * Nu) * (2 * Nv));
	saveGPUrealtxt(d_Filter,		"C:\\Users\\angelo\\Documents\\CEM\\ParticleSwarm\\ParticleSwarmSynthesis\\ParticleSwarmSynthesisMatlab\\d_Filter.txt",   (2 * Nu) * (2 * Nv));

	// --- Generating the (csi, eta) grid and the Legendre polynomials
	thrust::pair<thrust::pair<double *, double *>, double *> d_LegendreTuple = generateLegendreFactorized<double>(Num_unknowns_x, Num_unknowns_y, M_x, M_y);
	thrust::pair<double *, double *> d_CSI_ETA = d_LegendreTuple.first;
	double *d_CSI = d_CSI_ETA.first;
	double *d_ETA = d_CSI_ETA.second;
	double *d_LEG = d_LegendreTuple.second;
	
	// --- Generating the Zernike polynomials
	double *d_ZERNIKE = generateZernikep(d_CSI, d_ETA, Num_unknowns_phases, M_x, M_y);
	
	// --- Loading the masks
	double *d_External_Coverage = loadGPUrealtxt("C:\\Users\\angelo\\Documents\\CEM\\ParticleSwarm\\ParticleSwarmSynthesis\\ParticleSwarmSynthesisMatlab\\External_Coverage.txt", d_External_Coverage, (2 * Nu) * (2 * Nv));
	double *d_Internal_Coverage = loadGPUrealtxt("C:\\Users\\angelo\\Documents\\CEM\\ParticleSwarm\\ParticleSwarmSynthesis\\ParticleSwarmSynthesisMatlab\\Internal_Coverage.txt", d_Internal_Coverage, (2 * Nu) * (2 * Nv));

	/***********/
	/* TESTING */
	/***********/

	// --- Generating Zernike coefficients
	double *h_Coeff_Zernike = (double *)malloc(Num_unknowns_phases * sizeof(double)); 
	h_Coeff_Zernike[0] = -10.;
	h_Coeff_Zernike[1] =  50.;
	h_Coeff_Zernike[2] =   8.;
	h_Coeff_Zernike[3] =   9.;
	h_Coeff_Zernike[4] =   0.;
	h_Coeff_Zernike[5] =   0.;
	double *d_Coeff_Zernike;	gpuErrchk(hipMalloc(&d_Coeff_Zernike, Num_unknowns_phases * sizeof(double)));
	gpuErrchk(hipMemcpy(d_Coeff_Zernike, h_Coeff_Zernike, Num_unknowns_phases * sizeof(double), hipMemcpyHostToDevice));

	// --- Loading Lagrange coefficients
	double *d_Coeff_Lagrange_x = loadGPUrealtxt("C:\\Users\\angelo\\Documents\\Sintesi\\Sintesi_POS_Aperiodic_Reflectarray\\Coeff_legendre_x_init_vett.txt", d_Coeff_Lagrange_x, Num_unknowns_x * Num_unknowns_y);
	double *d_Coeff_Lagrange_y = loadGPUrealtxt("C:\\Users\\angelo\\Documents\\Sintesi\\Sintesi_POS_Aperiodic_Reflectarray\\Coeff_legendre_y_init_vett.txt", d_Coeff_Lagrange_y, Num_unknowns_x * Num_unknowns_y);
	
	// --- Calculate far field
	double2_ *d_far_field = raFarFieldCalculation((double *)d_Coeff_Zernike, (double *)d_ZERNIKE, 
											      (double *)d_Coeff_Lagrange_x, (double *)d_Coeff_Lagrange_y, (double *)d_LEG,   
											      (double *)d_U_discrete, (double *)d_V_discrete,
												  (double *)d_Filter,
												  Num_unknowns_phases, Num_unknowns_x, Num_unknowns_y,
												  handle, 
											      (double)feed_center_x, (double)feed_center_y, (double)feed_center_z,
												  (double)alfa, (double)beta, (double)mfact, 
												  (double)a_prime, (double)b_prime,
												  M_x, M_y,
												  Nu, Nv);

	saveGPUcomplextxt(d_far_field,	"C:\\Users\\angelo\\Documents\\CEM\\ParticleSwarm\\ParticleSwarmSynthesis\\ParticleSwarmSynthesisMatlab\\Far_Field_NUFFT.txt", (2 * Nu) * (2 * Nv));

	double Functional = raFunctionalCalculation(d_far_field, d_Internal_Coverage, d_External_Coverage, Nu, Nv);

	printf("Functional = %f\n", Functional);

	return 0;
}

